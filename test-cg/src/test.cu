#include "../include/spmm/spmm.cuh"
#include "../include/spmm/spmm_cg.cuh"
#include "../include/dataloader/dataloader.hpp"
#include "../include/util/ramArray.cuh"
#include "../include/util/check.cuh"

#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>         // cusparseSpGEMM
#include <fstream>
#include <iostream>
#include <stdio.h>  // printf
#include <stdlib.h> // EXIT_FAILURE


// None-template, No timer version
enum Mode {
    check,
    test,
    tune,
};

int main(int argc, const char **argv) {

    // Check command-line argument
    if (argc < 3) {
        printf("Require command-line argument: path of the sparse matrix file in "
                ".mtx format. Feature size. Mode.\n");
        return EXIT_FAILURE;
    }
    const char *filename = argv[1];
    int feature_size = atoi(argv[2]);
    Mode mode = static_cast<Mode>(atoi(argv[3]));

    // Load Sparse Matrix
    SpMatCsrDescr_t<Index, DType> H = SingleDataLoader<Index, DType>(filename);
    int nrow = H.nrow;
    int ncol = H.ncol;

    // Prepare Dense Matrix
    util::RamArray<DType> in_feature(ncol * feature_size), out_feature(nrow * feature_size), out_ref(nrow * feature_size);
    in_feature.fill_default_one();
    out_feature.fill_zero_h();
    out_ref.fill_zero_h();

    // CopyToDevice
    H.upload();
    in_feature.upload();
    out_feature.upload();
    out_ref.upload();

    // Call the kernels
    if (mode == Mode::check) {
        util::spmm_reference_host<Index, DType>(
            H.nrow, feature_size, H.sp_csrptr.h_array.get(),
            H.sp_csrind.h_array.get(), H.sp_data.h_array.get(),
            in_feature.h_array.get(), out_ref.h_array.get());

        checkSpMMErrorCG(csrspmm_parreduce_nnzbalance_cg);
    } else if (mode == Mode::test) {
        csrspmm_parreduce_nnzbalance<Index,DType>(H, feature_size, in_feature.d_array.get(),out_feature.d_array.get());
    }
    else {
        std::cout<<"Not implemented yet!"<<std::endl;
    }


    return 0;
}